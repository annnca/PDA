
#include <stdio.h>
#include<iostream>
#include "hip/hip_runtime.h"

using namespace std;

#define N 	4
#define INF 9999

__global__ void Floyd(int** graph, int k)
{
	int i = threadIdx.x;
	for(int j = 0; j <= N; j++){
		if (graph[i][k] + graph[k][j] < graph[i][j])
			graph[i][j] = graph[i][k] + graph[k][j];
	}
}


int main()
{
	int h_graph[N][N] = {
		0,5,9999, 10,
		9999, 0,3, 9999,
		9999, 9999, 0,1,
		9999, 9999, 9999,0
	};
	
	size_t size = N * N * sizeof(int);

	int** d_graph;
	
	hipMalloc(&d_graph, size);

	hipMemcpy(d_graph, h_graph, size, hipMemcpyHostToDevice);

	int numBlocks = 1;
	dim3 threadsPerBlock(N);

	for (int k = 0; k < N; k++) 
	{
		Floyd<<<numBlocks, threadsPerBlock>>>(d_graph, k);
	}

	hipMemcpy(h_graph, d_graph, size, hipMemcpyDeviceToHost);

	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			if (h_graph[i][j] == INF){
				cout << "INF ";
			}
			else{
				cout << h_graph[i][j] << " ";
			}
		}
		cout << endl;
	}
	
	hipFree(d_graph);
}